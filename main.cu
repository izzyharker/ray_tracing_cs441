#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <fstream>
#include <float.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"
#include "light.h"


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray& r, hitable **world, spotlight ** light, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 30; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            vec3 l = (*light)->hit(world, rec, cur_ray);
            if(rec.mat->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation*l;
                cur_ray = scattered;
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return cur_attenuation; // exceeded recursion
}

__global__ void render_init(int nx, int ny, hiprandState *state) {
    // set up random values for pixels
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= nx) || (j >= ny)) return;
    int pixel_index = j*nx + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &state[pixel_index]);
}

// render
__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable ** world, spotlight **light, hiprandState *state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;

    // antialiasing - send 4 random rays through each pixel on the screen and average their colors
    hiprandState local_rand_state = state[pixel_index];
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, light, &local_rand_state);
    }

    state[pixel_index] = local_rand_state;

    // calculate color
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);

    fb[pixel_index] = col;
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera ** cam, spotlight **light, int nx, int ny, int num) {
    float r = cos(M_PI/4);
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        // d_list[0] = new sphere(vec3(0,0,-1), 0.5,
        //                        new lambertian(vec3(0.1, 0.2, 0.5)));
        d_list[0] = new sphere(vec3(0,-100.5,-1), 100,
                               new lambertian(vec3(0.8, 0.8, 0.0)));
        // d_list[2] = new sphere(vec3(1,0,-1), 0.5,
        //                        new metal(vec3(0.8, 0.6, 0.2), 0.2));
        d_list[1] = new sphere(vec3(-1,0,-1), .5,
                               new dielectric(1.5));
        // d_list[2] = new sphere(vec3(-1,0,-1), -0.45,
        //                        new dielectric(1.5));
        *d_world  = new hitable_list(d_list,num);

        // set up vectors for camera
        vec3 lookfrom(-2, 2, 1);
        vec3 lookat(0, 0, -1);
        vec3 vup(0, 1, 0);
        float vfov = 60;
        float aspect = float(nx)/float(ny);
        float aperture = 0.1;
        float focus = (lookfrom - lookat).length();

        *cam = new camera(lookfrom, lookat, vup, vfov, aspect, aperture, focus);

        *light = new spotlight(vec3(2, 2, 1), lookat, 45, .2, 1);
    }
}

__global__ void free_world(hitable **d_list, hitable ** d_world, camera ** cam) {
    delete *(d_list);
    delete *(d_list + 1);
    delete *d_world;
    delete *cam;
}

void write_image(std::string filename, vec3 *fb, int nx, int ny) {
    std::ofstream f;
    f.open(filename);
    f << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            float r = fb[pixel_index][0];
            float g = fb[pixel_index][1];
            float b = fb[pixel_index][2];
            int ir = min(255, int(255.99*r));
            int ig = min(255, int(255.99*g));
            int ib = min(255, int(255.99*b));
            f << ir << " " << ig << " " << ib << "\n";
        }
    }
    f.close();
}

int main() {
    int nx = 600;
    int ny = 300;
    int ns = 10;
    // int nx = 2000;
    // int ny = 1000;
    // int ns = 100;
    int tx = 16;
    int ty = 32;

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);
 
    // allocate fb = buffer for image
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // make world
    hitable **d_list;
    int num_hitables = 2;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));

    camera **cam;
    checkCudaErrors(hipMalloc((void **)&cam, sizeof(camera *)));

    spotlight **light;
    checkCudaErrors(hipMalloc((void **)&light, sizeof(spotlight *)));

    create_world<<<1,1>>>(d_list,d_world, cam, light, nx, ny, num_hitables);


    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(nx/tx+1, ny/ty+1);
    dim3 threads(tx, ty);

    // init random values - for antialiasing
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));

    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);

    render<<<blocks, threads>>>(fb, nx, ny, ns, cam, d_world, light, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::string filename = "out.ppm";
    write_image(filename, fb, nx, ny);


    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world, cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();

    return 0;
}